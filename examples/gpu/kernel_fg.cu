#include <hip/hip_runtime.h>

/*
 * Kernel for evaluation of f and g.
 *
 * This takes in vectors u & v of length N, and returns f & g.
 *
 */
__global__ void kernel_fg(double * f, double * g, const double * u, const double * v, const int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N){
    f[idx] = 0.0;   
    f[idx] += -u[idx]*v[idx]*v[idx] + 0.054*(1-u[idx]);
    g[idx] = 0.0;
    g[idx] += u[idx]*v[idx]*v[idx] - (0.054+0.063)*v[idx];
  }
}


