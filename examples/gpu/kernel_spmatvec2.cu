#include <hip/hip_runtime.h>

/*
 * Kernel for Sparse Matrix Vector Multiply
 *
 * This takes an N x N sparse matrix and multiplies it by an N-vector x.
 *
 * The matrix has exactly M entries per row (some could be zero) and is stored
 * in the 2D matrix As and Aj which contain the coefficients and column numbers
 * respectively.  Aj is assumed to count from 1 not zero (e.g., it comes from Matlab)
 * The result is returned in y.
 *
 * Colin Macdonald, 2013-01-13
 */
__global__ void kernel_spmatvec2(double * y, const double * As, const int * Aj,
                                 const double * x, const int N, const unsigned int M)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int k;
  unsigned int j;
  if (idx < N) {
    y[idx] = 0.0;   // todo: any advantage to adding into a temporary double?
    for (j = 0; j < M; j++) {
      k = Aj[idx + j*N] - 1;  // -1 here for matlab indexing
      y[idx] += As[idx + j*N] * x[k];
    }
  }
}

